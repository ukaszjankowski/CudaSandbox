#include "hip/hip_runtime.h"
// Image processing in NVIDIA CUDA
// Copyright 2016 by �ukasz Jankowski

#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

// RGB To Greenscale conversion factors
#define RED_FACTOR		0.299
#define GREEN_FACTOR	0.587
#define BLUE_FACTOR		0.114

__global__ void GrayscaleKernel(pixel *data)
{
    int i = threadIdx.x;
	data[i].gray = RED_FACTOR   * data[i].r 
				 + GREEN_FACTOR * data[i].g 
				 + BLUE_FACTOR  * data[i].b;
}

__global__ void InvertKernel(pixel *data)
{
	int i = threadIdx.x;
	data[i].r = 255 - data[i].r;
	data[i].g = 255 - data[i].g;
	data[i].b = 255 - data[i].b;
}

__global__ void ContrastKernel(pixel *data, float contrast)
{
	if (contrast < 0) contrast = 0;
	if (contrast > 2) contrast = 2;

	int i = threadIdx.x;
	data[i].r = contrast * data[i].r < data[i].r ? 255 : contrast * data[i].r;
	data[i].g = contrast * data[i].g < data[i].g ? 255 : contrast * data[i].g;
	data[i].b = contrast * data[i].b < data[i].b ? 255 : contrast * data[i].b;
}

int main()
{
    const int arraySize = 1024;
	pixel *data = (pixel*)malloc(arraySize * sizeof(pixel));

	hipError_t cudaStatus;

	for (long i = 0; i < arraySize; i++) {
		data[i].gray = 0;
		data[i].r = 10;
		data[i].g = 20;
		data[i].b = 30;
	}

	time_t cudaStartTime = time(NULL);
	cudaStatus = Grayscale(data, arraySize);
	time_t cudaEndTime = time(NULL);
	printf("GPU: %i\n", cudaEndTime - cudaStartTime);

	time_t cpuStartTime = time(NULL);
	for (size_t i = 0; i < 1000000; i++) {
		for (size_t j = 0; j < arraySize; j++)
		{
			data[j].gray = RED_FACTOR   * data[j].r
						 + GREEN_FACTOR * data[j].g
						 + BLUE_FACTOR  * data[j].b;
		}
	}
	time_t cpuEndTime = time(NULL);
	printf("CPU (single thread): %i\n", cpuEndTime - cpuStartTime);

	//printf("Pixel: %d\n", data[0].gray);

	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Cuda failed!");
		getchar();
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		getchar();
        return 1;
    }

	getchar();
    return 0;
}

__declspec(dllexport) hipError_t Grayscale(pixel *data, unsigned int size)
{
	pixel *dev_data = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) return CudaFail("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n", data, cudaStatus);

	cudaStatus = hipMalloc((void**)&dev_data, size * sizeof(pixel));
	if (cudaStatus != hipSuccess) return CudaFail("hipMalloc failed!\n", data, cudaStatus);

	cudaStatus = hipMemcpy(dev_data, data, size * sizeof(pixel), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) return CudaFail("hipMemcpy failed!\n", data, cudaStatus);

    // Launch a kernel on the GPU with one thread for each element.
	for (int i = 0; i < 1000000; i++) {
		GrayscaleKernel<<<1, size >>>(dev_data);
	}

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) return CudaFail("Kernel launch failed\n", data, cudaStatus);
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) return CudaFail("hipDeviceSynchronize returned error after launching Kernel!\n", data, cudaStatus);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(data, dev_data, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) return CudaFail("hipMemcpy failed!\n", data, cudaStatus);

    hipFree(dev_data);
    return cudaStatus;
}

hipError_t CudaFail(const char *message, pixel *data, hipError_t status) {
	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", status);
	hipFree(data);
	return status;
}